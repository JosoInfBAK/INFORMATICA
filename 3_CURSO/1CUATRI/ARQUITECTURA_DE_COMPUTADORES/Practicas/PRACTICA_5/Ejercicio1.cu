#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n, unsigned int *Nth)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    Nth[0] = stride * gridDim.x;
    printf("Thread index: %d From block number: %d\n", index, blockIdx.x);
    for (int i = index; i < n; i += stride)
    {
        out[i] = a[i] + b[i];
    }
}
int main(int argc, char **argv)
{
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    unsigned int *d_nth;
    int manBlockID;
    if (argc < 2)
    {
        printf("Utilizacion del programa : './ejecutable parametro'\nSiendo parametro :\n 0 : Numero de bloques = 10000000/256 (default) y 256 Hilos por bloque\n 1 : Entrada de Numero de bloques y numero de hilos por bloque manual\n");
        exit(0);
    }
    else
    {
        // Allocate host memory
        a = (float *)malloc(sizeof(float) * N);
        b = (float *)malloc(sizeof(float) * N);
        out = (float *)malloc(sizeof(float) * N);
        unsigned int *T_NThreads;
        int blockID;
        int nThreads;
        if (atoi(argv[1]) == 0)
        {
            blockID = N / 256;
            nThreads = 256;
        }
        else if (atoi(argv[1]) == 1)
        {
            printf("Ingrese el numero de bloques : ");
            scanf("%d", &manBlockID);
            blockID = manBlockID;
            printf("Ingrese el numero de hilos por bloque : ");
            scanf("%d", &nThreads);
        }
        else
        {
            printf("Parametro invalido\n");
            exit(0);
        }
        // Initialize host arrays
        for (int i = 0; i < N; i++)
        {
            a[i] = 1.0f;
            b[i] = 2.0f;
        }
        // Allocate device memory
        hipMalloc((void **)&d_a, sizeof(float) * N);
        hipMalloc((void **)&d_b, sizeof(float) * N);
        hipMalloc((void **)&d_out, sizeof(float) * N);
        hipMalloc((void **)&d_nth, sizeof(int));

        // Transfer data from host to device memory
        hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
        // Executing kernel
        vector_add<<<blockID, nThreads>>>(d_out, d_a, d_b, N, d_nth);
        // Transfer data back to host memory
        hipMemcpy(out, d_out, sizeof(float) * N,
                   hipMemcpyDeviceToHost);
        hipMemcpy(&T_NThreads, d_nth, sizeof(int),
                   hipMemcpyDeviceToHost);
        printf("Number of Blocks: %d \n", blockID);
        printf("Number of Threads per block: %d \n", nThreads);
        printf("Total number of threads: %x \n", T_NThreads[0]);
        // Verification
        for (int i = 0; i < N; i++)
        {
            assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
        }
        printf("PASSED\n");
        // Deallocate device memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_out);
        // Deallocate host memory
        free(a);
        free(b);
        free(out);
    }
}